#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Function to measure time using CUDA events
void measureTimeCUDAEvent(hipEvent_t start, hipEvent_t stop) {
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time Elapsed: %f ms\n", elapsedTime);
}


// Initialize Ising model with random initial state
void initialize(int* lattice, int n) {
    srand(time(NULL));
    // Random +1 or -1
    for (int i = 0; i < n * n; i++) {
        lattice[i] = rand() % 2 * 2 - 1;
    }
}

// Print the current state of the Ising model
void printLattice(int* lattice, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%2d ", lattice[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Perform Ising model simulation
__global__ void simulate(int* current, int* next, int n, int BLOCK_WIDTH, int TILE_WIDTH) {

    extern __shared__ int mem[];

    int MEM_WIDTH = BLOCK_WIDTH + 2;
    
    int k = blockIdx.y * BLOCK_WIDTH + threadIdx.y * TILE_WIDTH;
    int l = blockIdx.x * BLOCK_WIDTH + threadIdx.x * TILE_WIDTH;
    
    for (int i = k; i < k + TILE_WIDTH && i < (blockIdx.y + 1) * BLOCK_WIDTH && i < n; i++){
        for (int j = l; j < l + TILE_WIDTH && j < (blockIdx.x + 1) * BLOCK_WIDTH && j < n; j++){

            int memY = i % BLOCK_WIDTH;
            int memX = j % BLOCK_WIDTH;

            int memIdx = (memY + 1) * MEM_WIDTH  + (memX + 1);


            mem[memIdx] = current[i * n + j];

            // Load additional boundary data to shared memory
            if (memY == 0) {
                mem[memIdx - MEM_WIDTH] = current[((i - 1 + n) % n) * n + j];
            }

            if (memY == BLOCK_WIDTH - 1 || i == n - 1) {
                mem[memIdx + MEM_WIDTH] = current[((i + 1) % n) * n + j];
            }

            if (memX == 0) {
                mem[memIdx - 1] = current[i * n + (j - 1 + n) % n];
            }

            if (memX == BLOCK_WIDTH - 1  || j == n - 1) {
                mem[memIdx + 1] = current[i * n + (j + 1) % n];
            }

            // FOR CORRECTION
            // printf("Block: %d,%d - Thread: %d,%d - i: %d, j: %d\nUp s: %d, c: %d\nDown s: %d, c: %d\nLeft s: %d, c: %d\nRight s: %d, c: %d\nCenter s: %d, c: %d\n\n", 
            //     blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, i, j, 
            //     mem[memIdx - MEM_WIDTH], current[((i - 1 + n) % n) * n + j],
            //     mem[memIdx + MEM_WIDTH], current[((i + 1) % n) * n + j],
            //     mem[memIdx - 1], current[i * n + (j - 1 + n) % n],
            //     mem[memIdx + 1], current[i * n + (j + 1) % n],
            //     mem[memIdx], current[i * n + j]
            // );
        }
    }

    __syncthreads();

    for (int i = k; i < k + TILE_WIDTH && i < (blockIdx.y + 1) * BLOCK_WIDTH && i < n; i++){
        for (int j = l; j < l + TILE_WIDTH && j < (blockIdx.x + 1) * BLOCK_WIDTH && j < n; j++){

            int memY = i % BLOCK_WIDTH;
            int memX = j % BLOCK_WIDTH;

            int memIdx = (memY + 1) * MEM_WIDTH  + (memX + 1);

            int sum = mem[memIdx - MEM_WIDTH] +
                    mem[memIdx + MEM_WIDTH] +
                    mem[memIdx - 1] +
                    mem[memIdx + 1] +
                    mem[memIdx];
            
            next[i * n + j] = (sum > 0) - (sum < 0);
        }
    }

            

}

void saveArrayToFile(int* array, int size, const char* filename) {
    FILE* file = fopen(filename, "wb");

    if (file != NULL) {
        fwrite(array, sizeof(int), size, file);

        fclose(file);

    } else {
        fprintf(stderr, "Error opening file: %s\n", filename);
    }
}

int main() {
    int n, k;

    int TILE_WIDTH, BLOCK_WIDTH;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    

    // Get the size and number of iters
    printf("Enter the width of the Ising model (n): ");
    scanf("%d", &n);
    printf("Enter the number of iterations (k): ");
    scanf("%d", &k);
    printf("Define Block Width (BLOCK_WIDTH): ");
    scanf("%d", &BLOCK_WIDTH);
    printf("Define Tile Width (TILE_WIDTH): ");
    scanf("%d", &TILE_WIDTH);
    

    size_t size = n * n * sizeof(int);

    // Mem allocation
    int* lattice1 = (int*)malloc(size);
    int* lattice2 = (int*)malloc(size);


    // Initialize Ising Model 
    initialize(lattice1, n);
    printf("\nInitial state:\n");
    //printLattice(lattice1, n);


    int *d_lattice1, *d_lattice2;
    hipMalloc(&d_lattice1, size);
    hipMalloc(&d_lattice2, size);

    hipMemcpy(d_lattice1, lattice1, size, hipMemcpyHostToDevice);

    int memSize = (BLOCK_WIDTH + 2) * (BLOCK_WIDTH + 2) * sizeof(int);

    dim3 dimBlock(ceil(BLOCK_WIDTH / (float)TILE_WIDTH), ceil(BLOCK_WIDTH / (float)TILE_WIDTH), 1);
    dim3 dimGrid(ceil(n / (float)BLOCK_WIDTH), ceil(n / (float)BLOCK_WIDTH), 1);

    // Record the start event
    hipEventRecord(start, 0);

    for (int i = 0; i < k; i++) {
        //printf("Iteration %d:\n", i);
        //printLattice(lattice1, n);

        simulate<<<dimGrid, dimBlock, memSize>>>(d_lattice1, d_lattice2, n, BLOCK_WIDTH, TILE_WIDTH);

        // Swap lattices
        int* temp = d_lattice1;
        d_lattice1 = d_lattice2;
        d_lattice2 = temp;

        hipDeviceSynchronize();
    }
    // Record the stop event
    hipEventRecord(stop, 0);

    // Measure and print the execution time
    measureTimeCUDAEvent(start, stop);

    hipMemcpy(lattice1, d_lattice1, size, hipMemcpyDeviceToHost);
    hipMemcpy(lattice2, d_lattice2, size, hipMemcpyDeviceToHost);

    // Final state
    printf("\nFinal state:\n");
    //printLattice(lattice1, n);

    // Save array to file
    saveArrayToFile(lattice1, n * n, "v3_2.bin");

    // Cleanup
    free(lattice1);
    free(lattice2);

    hipFree(d_lattice1);
    hipFree(d_lattice2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}