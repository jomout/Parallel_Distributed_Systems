#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>


// Μeasure time using CUDA events
void measureTimeCUDAEvent(hipEvent_t start, hipEvent_t stop) {
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time Elapsed: %f ms\n", elapsedTime);
}

// Initialize Ising model with random initial state
void initialize(int* lattice, int n) {
    srand(time(NULL));
    // Random +1 or -1
    for (int i = 0; i < n * n; i++) {
        lattice[i] = rand() % 2 * 2 - 1;
    }
}

// Print the current state of the Ising model
void printLattice(int* lattice, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%2d ", lattice[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Perform Ising model simulation
__global__ void simulate(int* current, int* next, int n) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i >= n) || (j >= n))
        return;
    
    int sum = current[((i - 1 + n) % n) * n + j] +
                        current[((i + 1) % n) * n + j] +
                        current[i * n + j] +
                        current[i * n + (j - 1 + n) % n] +
                        current[i * n + (j + 1) % n];

    // FOR CORRECTION
    // printf("Block: %d,%d - Thread: %d,%d - i: %d, j: %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, i, j);

    next[i * n + j] = (sum > 0) - (sum < 0);
}

// Saving array for correction later
void saveArrayToFile(int* array, int size, const char* filename) {
    FILE* file = fopen(filename, "wb");

    if (file != NULL) {
        fwrite(array, sizeof(int), size, file);

        fclose(file);

    } else {
        fprintf(stderr, "Error opening file: %s\n", filename);
    }
}


int main() {
    int n, k;

    int BLOCK_WIDTH;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    

    // Get the size and number of iters
    printf("Enter the width of the Ising model (n): ");
    scanf("%d", &n);
    printf("Enter the number of iterations (k): ");
    scanf("%d", &k);
    printf("Define Block Width (BLOCK_WIDTH): ");
    scanf("%d", &BLOCK_WIDTH);


    size_t size = n * n * sizeof(int);

    // Memery allocation
    int* lattice1 = (int*)malloc(size);
    int* lattice2 = (int*)malloc(size);


    // Initialize Ising Model 
    initialize(lattice1, n);
    printf("\nInitial state:\n");
    //printLattice(lattice1, n);

    // GPU arrays
    int *d_lattice1, *d_lattice2;
    hipMalloc(&d_lattice1, size);
    hipMalloc(&d_lattice2, size);

    hipMemcpy(d_lattice1, lattice1, size, hipMemcpyHostToDevice);


    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
    dim3 dimGrid(ceil(n / (float)BLOCK_WIDTH), ceil(n / (float)BLOCK_WIDTH), 1);


    // Record the start event
    hipEventRecord(start, 0);

    for (int i = 0; i < k; i++) {
        //printf("Iteration %d:\n", i);
        //printLattice(lattice1, n);

        simulate<<<dimGrid, dimBlock>>>(d_lattice1, d_lattice2, n);

        // Swap lattices
        int* temp = d_lattice1;
        d_lattice1 = d_lattice2;
        d_lattice2 = temp;

        hipDeviceSynchronize();
    }
    
    // Record the stop event
    hipEventRecord(stop, 0);

    // Measure and print the execution time
    measureTimeCUDAEvent(start, stop);

    hipMemcpy(lattice1, d_lattice1, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(lattice2, d_lattice2, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // Final state
    printf("\nFinal state:\n");
    //printLattice(lattice1, n);

    // Save array to file
    saveArrayToFile(lattice1, n * n, "v1.bin");

    // Cleanup
    free(lattice1);
    free(lattice2);

    hipFree(d_lattice1);
    hipFree(d_lattice2);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}